#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void hello_world() {
    printf("Hello, World\n");
}

int main() {
    hello_world<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}